
#include "hip/hip_runtime.h"

#include "hipblas.h"

#include <stdio.h>
#include <iostream>


const int n = 6;


int main()
{
	hipblasHandle_t handle;
	
	float *x = new float[n];
	float *y = new float[n];


	for (int i = 0; i < n; i++)
	{
		x[i] = float(i);
		y[i] = float(i);
	}


	float *d_x, *d_y;

	hipMalloc((void**)&d_x, n*sizeof(float));
	hipMalloc((void**)&d_y, n*sizeof(float));

	hipblasCreate(&handle);

	hipblasSetVector(n, sizeof(float), x, 1, d_x, 1);
	hipblasSetVector(n, sizeof(float), y, 1, d_y, 1);

	float result;

	hipblasSdot(handle, n, d_x, 1, d_y, 1, &result);
	
	std::cout << "The Dot Product of x and y is :" << result << std::endl;

	hipFree(d_x); hipFree(d_y);
	hipblasDestroy(handle);
	delete[] x; delete[] y;
	return 0;

    return 0;
}
