#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>

int main()
{
	hipblasHandle_t handle;

	hipblasCreate(&handle);

	hipblasDestroy(handle);

    return 0;
}

