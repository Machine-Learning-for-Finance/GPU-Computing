
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>


int main()
{
	int numDev = 0;
	hipGetDeviceCount(&numDev);
	std::cout << "There are " << numDev << " GPU's in this system." << std::endl;

	hipSetDevice(0);

	hipDeviceEnablePeerAccess(1, 0);

	hipSetDevice(1);

	hipDeviceEnablePeerAccess(0, 0);

	/*
	
	Run Code Using Both
	
	*/
	hipSetDevice(0);
	hipDeviceDisablePeerAccess(1);
	
	hipSetDevice(1);
	hipDeviceDisablePeerAccess(0);



    return 0;
}
